
/**
*    ICRAR - International Centre for Radio Astronomy Research
*    (c) UWA - The University of Western Australia
*    Copyright by UWA (in the framework of the ICRAR)
*    All rights reserved
*
*    This library is free software; you can redistribute it and/or
*    modify it under the terms of the GNU Lesser General Public
*    License as published by the Free Software Foundation; either
*    version 2.1 of the License, or (at your option) any later version.
*
*    This library is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
*    Lesser General Public License for more details.
*
*    You should have received a copy of the GNU Lesser General Public
*    License along with this library; if not, write to the Free Software
*    Foundation, Inc., 59 Temple Place, Suite 330, Boston,
*    MA 02111-1307  USA
*/

#include <icrar/leap-accelerate/math/cuda/matrix_multiply.h>

#include <icrar/leap-accelerate/cuda/helper_cuda.cuh>
#include <hipblas.h>
#include <hipblaslt.h>

#include <type_traits>

template<typename T>
struct is_cublas_supported : public std::false_type {};
template<>
struct is_cublas_supported<double> : public std::true_type {};
template<>
struct is_cublas_supported<float> : public std::true_type {};
template<>
struct is_cublas_supported<int32_t> : public std::true_type {};

namespace icrar
{
namespace cuda
{
    /**
     * @brief Performs matrix multiplcation with offset of the form C = A * B
     */
    template<typename T, typename=std::enable_if_t<is_cublas_supported<T>::value>>
    __host__ void mat_mul(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const T* A, const T* B, T* C)
    {
        const double alpha = 1.0;
        const double beta = 0.0;
        hipblasOperation_t transa = hipblasOperation_t::HIPBLAS_OP_N;
        hipblasOperation_t transb = hipblasOperation_t::HIPBLAS_OP_N;

        int lda = m;
        int ldb = k;
        int ldc = m;

        hipblasComputeType_t computeType;
        hipDataType dataType;
        if(std::is_same<T, double>::value)
        {
            computeType = HIPBLAS_COMPUTE_64F;
            dataType = HIP_R_64F;
        }
        else if(std::is_same<T, float>::value)
        {
            computeType = HIPBLAS_COMPUTE_32F;
            dataType = HIP_R_32F;
        }
        else if(std::is_same<T, std::int32_t>::value)
        {
            computeType = HIPBLAS_COMPUTE_32I;
            dataType = HIP_R_32I;
        }
        else
        {
            throw invalid_argument_exception("invalid template", "T", __FILE__, __LINE__);
        }

        checkCudaErrors(hipblasGemmEx(
            handle,
            transa, 
            transb,
            m, n, k,
            &alpha,
            A, dataType, lda,
            B, dataType, ldb,
            &beta,
            C, dataType, ldc,
            computeType,
            hipblasGemmAlgo_t::CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }

    /**
     * @brief Performs matrix multiplcation with offset of the form C = (A * B) + C
     */
    template<typename T, typename=std::enable_if_t<is_cublas_supported<T>::value>>
    __host__ void mat_mul_add(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const T* A, const T* B, T* C)
    {
        const double alpha = 1.0;
        const double beta = 1.0;
        hipblasOperation_t transa = hipblasOperation_t::HIPBLAS_OP_N;
        hipblasOperation_t transb = hipblasOperation_t::HIPBLAS_OP_N;

        int lda = m;
        int ldb = k;
        int ldc = m;

        hipblasComputeType_t computeType;
        hipDataType dataType;
        if(std::is_same<T, double>::value)
        {
            computeType = HIPBLAS_COMPUTE_64F;
            dataType = HIP_R_64F;
        }
        else if(std::is_same<T, float>::value)
        {
            computeType = HIPBLAS_COMPUTE_32F;
            dataType = HIP_R_32F;
        }
        else if(std::is_same<T, std::int32_t>::value)
        {
            computeType = HIPBLAS_COMPUTE_32I;
            dataType = HIP_R_32I;
        }
        else
        {
            throw invalid_argument_exception("invalid template", "T", __FILE__, __LINE__);
        }

        checkCudaErrors(hipblasGemmEx(
            handle,
            transa, 
            transb,
            m, n, k,
            &alpha,
            A, dataType, lda,
            B, dataType, ldb,
            &beta,
            C, dataType, ldc,
            computeType,
            hipblasGemmAlgo_t::CUBLAS_GEMM_DEFAULT_TENSOR_OP));
    }

    template<typename T>
    __host__ void mat_mul(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const T* A, const T* B, T* C)
    {
        hipblasOperation_t transa = hipblasOperation_t::HIPBLAS_OP_N;
        hipblasOperation_t transb = hipblasOperation_t::HIPBLAS_OP_N;

        size_t lda = m;
        size_t ldb = k;
        size_t ldc = m;

        const double alpha = 1.0;
        const double beta = 1.0;

        hipblasLtMatmulDescOpaque_t operationDesc = {};
        hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
        hipblasLtMatmulAlgo_t algo = {};

        const int32_t algoId = 10;
        const cublasLtMatmulTile_t tileId = CUBLASLT_MATMUL_TILE_16x16;
        const cublasLtReductionScheme_t reductionMode = CUBLASLT_REDUCTION_SCHEME_INPLACE;
        const int32_t splitKFactor = 256;

        // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
        // set the transforms for A and B

        hipblasComputeType_t computeType;
        hipDataType dataType;
        if(std::is_same<T, double>::value)
        {
            computeType = HIPBLAS_COMPUTE_64F;
            dataType = HIP_R_64F;
        }
        else if(std::is_same<T, float>::value)
        {
            computeType = HIPBLAS_COMPUTE_32F;
            dataType = HIP_R_32F;
        }
        else if(std::is_same<T, std::int32_t>::value)
        {
            computeType = HIPBLAS_COMPUTE_32I;
            dataType = HIP_R_32I;
        }
        else
        {
            throw invalid_argument_exception("invalid template", "T", __FILE__, __LINE__);
        }

        checkCudaErrors(cublasLtMatmulDescInit(&operationDesc, computeType, dataType));
        checkCudaErrors(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
        checkCudaErrors(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));

        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCudaErrors(cublasLtMatrixLayoutInit(&Adesc, dataType, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
        checkCudaErrors(cublasLtMatrixLayoutInit(&Bdesc, dataType, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
        checkCudaErrors(cublasLtMatrixLayoutInit(&Cdesc, dataType, m, n, ldc));

        checkCudaErrors(cublasLtMatmulAlgoInit(
            handle,
            computeType, // compute
            dataType, //scale
            dataType, // A
            dataType, // B
            dataType, // C
            dataType, // D
            algoId,
            &algo));

        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tileId, sizeof(tileId)));
        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &reductionMode, sizeof(reductionMode)));
        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitKFactor, sizeof(splitKFactor)));

        size_t workspaceSize = 4 * 1024 * 1024;
        void *workspace = nullptr;
        checkCudaErrors(hipMalloc(&workspace, workspaceSize));

        hipStream_t stream = nullptr;

        checkCudaErrors(hipblasLtMatmul(
            handle,
            &operationDesc,
            &alpha,
            (void*)A,
            &Adesc,
            (void*)B,
            &Bdesc,
            &beta,
            (void*)C,
            &Cdesc,
            (void*)C,
            &Cdesc,
            &algo,
            (void*)workspace,
            workspaceSize,
            stream));

        checkCudaErrors(hipFree(workspace));
    }

    /**
     * @brief Performs matrix multiplcation with offset of the form D = (A * B) + C 
     * 
     * @tparam T 
     * @param handle 
     * @param m 
     * @param n 
     * @param k 
     * @param A 
     * @param B 
     * @param C 
     * @param D 
     * @return __host__ 
     */
    template<typename T>
    __host__ void mat_mul_add(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const T* A, const T* B, const T* C, T* D)
    {
        hipblasOperation_t transa = hipblasOperation_t::HIPBLAS_OP_N;
        hipblasOperation_t transb = hipblasOperation_t::HIPBLAS_OP_N;

        size_t lda = m;
        size_t ldb = k;
        size_t ldc = m;

        const double alpha = 1.0;
        const double beta = 1.0;

        hipblasLtMatmulDescOpaque_t operationDesc = {};
        hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
        hipblasLtMatmulAlgo_t algo = {};

        const int32_t algoId = 10;
        const cublasLtMatmulTile_t tileId = CUBLASLT_MATMUL_TILE_16x16;
        const cublasLtReductionScheme_t reductionMode = CUBLASLT_REDUCTION_SCHEME_INPLACE;
        const int32_t splitKFactor = 256;

        // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
        // set the transforms for A and B

        hipblasComputeType_t computeType;
        hipDataType dataType;
        if(std::is_same<T, double>::value)
        {
            computeType = HIPBLAS_COMPUTE_64F;
            dataType = HIP_R_64F;
        }
        else if(std::is_same<T, float>::value)
        {
            computeType = HIPBLAS_COMPUTE_32F;
            dataType = HIP_R_32F;
        }
        else if(std::is_same<T, std::int32_t>::value)
        {
            computeType = HIPBLAS_COMPUTE_32I;
            dataType = HIP_R_32I;
        }
        else
        {
            throw invalid_argument_exception("invalid template", "T", __FILE__, __LINE__);
        }

        //LtSgemm

        checkCudaErrors(cublasLtMatmulDescInit(&operationDesc, computeType, dataType));
        checkCudaErrors(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
        checkCudaErrors(hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));

        // create matrix descriptors, we are good with the details here so no need to set any extra attributes
        checkCudaErrors(cublasLtMatrixLayoutInit(&Adesc, dataType, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
        checkCudaErrors(cublasLtMatrixLayoutInit(&Bdesc, dataType, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
        checkCudaErrors(cublasLtMatrixLayoutInit(&Cdesc, dataType, m, n, ldc));

        checkCudaErrors(cublasLtMatmulAlgoInit(
            handle,
            computeType, // compute
            dataType, //scale
            dataType, // A
            dataType, // B
            dataType, // C
            dataType, // D
            algoId,
            &algo));

        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tileId, sizeof(tileId)));
        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &reductionMode, sizeof(reductionMode)));
        checkCudaErrors(cublasLtMatmulAlgoConfigSetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &splitKFactor, sizeof(splitKFactor)));

        size_t workspaceSize = 4 * 1024 * 1024;
        void *workspace = nullptr;
        checkCudaErrors(hipMalloc(&workspace, workspaceSize));

        hipStream_t stream = nullptr;

        checkCudaErrors(hipblasLtMatmul(
            handle,
            &operationDesc,
            &alpha,
            (void*)A,
            &Adesc,
            (void*)B,
            &Bdesc,
            &beta,
            (void*)C,
            &Cdesc,
            (void*)D,
            &Cdesc,
            &algo,
            (void*)workspace,
            workspaceSize,
            stream));

        checkCudaErrors(hipFree(workspace));
    }

    __host__ void mat_mul(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const double* A, const double* B, double* C)
    {
        mat_mul<double>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const float* A, const float* B, float* C)
    {
        mat_mul<float>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const int* A, const int* B, int* C)
    {
        mat_mul<int>(handle, m, n, k, A, B, C);
    }

    __host__ void mat_mul(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const double* A, const double* B, double* C)
    {
        mat_mul<double>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const float* A, const float* B, float* C)
    {
        mat_mul<float>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const int* A, const int* B, int* C)
    {
        mat_mul<int>(handle, m, n, k, A, B, C);
    }

    __host__ void mat_mul_add(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const double* A, const double* B, double* C)
    {
        mat_mul_add<double>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul_add(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const float* A, const float* B, float* C)
    {
        mat_mul_add<float>(handle, m, n, k, A, B, C);
    }
    __host__ void mat_mul_add(hipblasHandle_t handle, const size_t m, const size_t n, const size_t k, const int* A, const int* B, int* C)
    {
        mat_mul_add<int>(handle, m, n, k, A, B, C);
    }

    __host__ void mat_mul_add(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const double* A, const double* B, const double* C, double* D)
    {
        mat_mul_add<double>(handle, m, n, k, A, B, C, D);
    }
    __host__ void mat_mul_add(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const float* A, const float* B, const float* C, float* D)
    {
        mat_mul_add<float>(handle, m, n, k, A, B, C, D);
    }
    __host__ void mat_mul_add(hipblasLtHandle_t handle, const size_t m, const size_t n, const size_t k, const int* A, const int* B, const int* C, int* D)
    {
        mat_mul_add<int>(handle, m, n, k, A, B, C, D);
    }
} // namespace cuda
} // namespace icrar
