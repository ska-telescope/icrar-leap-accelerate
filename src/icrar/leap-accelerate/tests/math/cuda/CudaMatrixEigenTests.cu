#include "hip/hip_runtime.h"
/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 * MA 02111 - 1307  USA
 */

#include <hip/hip_runtime.h>

#include <icrar/leap-accelerate/tests/test_helper.h>
#include <icrar/leap-accelerate/tests/math/eigen_helper.h>
#include <icrar/leap-accelerate/cuda/helper_cuda.cuh>
#include <icrar/leap-accelerate/math/cuda/vector_eigen.cuh>
#include <icrar/leap-accelerate/math/cuda/matrix.h>

#include <icrar/leap-accelerate/math/cpu/matrix_invert.h>

#include <icrar/leap-accelerate/core/log/logging.h>

#include <Eigen/Core>

#include <gtest/gtest.h>

#include <stdio.h>
#include <array>

#include <icrar/leap-accelerate/common/eigen_stringutils.h>

namespace icrar
{
    class CudaMatrixEigenTests : public testing::Test
    {
        double TOLERANCE = 0.1;
        hipsolverHandle_t m_cusolverDnCtx;

    public:
        void SetUp() override
        {
            // See this page: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html
            int deviceCount = 0;
            checkCudaErrors(hipGetDeviceCount(&deviceCount));
            ASSERT_EQ(1, deviceCount);

            checkCudaErrors(hipsolverDnCreate(&m_cusolverDnCtx));
        }

        void TearDown() override
        {
            checkCudaErrors(hipsolverDnDestroy(m_cusolverDnCtx));
            checkCudaErrors(hipDeviceReset());
        }

        void TestVectorAdd()
        {
            constexpr int N = 10;
            auto a = Eigen::Matrix<double, N, 1>();
            a << 6,6,6,6,6, 6,6,6,6,6;

            auto b = Eigen::Matrix<double, N, 1>();
            b << 10,10,10,10,10, 10,10,10,10,10;

            auto c = Eigen::Matrix<double, N, 1>();

            icrar::cuda::h_add<double, N>(a, b, c);

            auto expected = Eigen::Matrix<double, N, 1>();
            expected << 16,16,16,16,16, 16,16,16,16,16;
            ASSERT_EQ(c, expected);
        }

        void TestPseudoInverse23(cuda::JobType jobType)
        {
            constexpr int M = 2;
            constexpr int N = 3;

            auto m1 = Eigen::MatrixXd(M, N);
            m1 <<
            1, 3, 5,
            2, 4, 6;

            auto m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType);
            ASSERT_MEQD(m1, m1 * m1d * m1, TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(2,2), m1 * m1d, TOLERANCE);
        }

        void TestPseudoInverse32Degenerate()
        {
            constexpr int M = 3;
            constexpr int N = 2;

            auto m1 = Eigen::MatrixXd(M, N);
            m1 <<
            0.5, 0.5,
            -1, -1,
            -0.5, -0.5;

            auto m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1);

            auto expected_m1d = Eigen::MatrixXd(N, M);
            expected_m1d <<
            0.166667, -0.333333, -0.166667,
            0.166667, -0.333333, -0.166667;

            ASSERT_MEQD(expected_m1d, m1d, TOLERANCE);
            ASSERT_MEQD(m1, m1 * m1d * m1, TOLERANCE);
        }

        void TestPseudoInverse33(cuda::JobType jobType)
        {
            constexpr int M = 3;
            constexpr int N = 3;

            auto m1 = Eigen::MatrixXd(M, N);
            m1 <<
            1, 2, 3,
            4, 5, 6,
            7, 8, 9;

            ASSERT_THROW(icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType), icrar::invalid_argument_exception);
            //auto m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType);
            //ASSERT_MEQD(m1, m1 * m1d * m1, TOLERANCE);
            //ASSERT_MEQD(Eigen::MatrixXd::Identity(3,3), m1 * m1d, TOLERANCE);
        }

        void TestPseudoInverse32(cuda::JobType jobType)
        {
            constexpr int M = 3;
            constexpr int N = 2;

            auto m1 = Eigen::MatrixXd(M, N);
            m1 <<
            1, 2,
            3, 4,
            5, 6;

            Eigen::MatrixXd m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType);
            ASSERT_MEQD(m1, m1 * (m1d * m1), TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(2,2), m1d * m1, TOLERANCE);
        }

        void TestPseudoInverse42(cuda::JobType jobType)
        {
            constexpr int M = 4;
            constexpr int N = 2;

            auto m1 = Eigen::MatrixXd(M, N);
            m1 <<
            1, 2,
            3, 4,
            5, 6,
            7, 8;

            Eigen::MatrixXd m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType);
            ASSERT_MEQD(m1, m1 * (m1d * m1), TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(2,2), m1d * m1, TOLERANCE);
        }

        void TestPseudoInverseMWA(cuda::JobType jobType)
        {
            constexpr int M = 8001;
            constexpr int N = 128;

            Eigen::MatrixXd m1 = Eigen::MatrixXd::Random(M, N);
            Eigen::MatrixXd m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, jobType);

            ASSERT_MEQD(m1, m1 * (m1d * m1), TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(N,N), m1d * m1, TOLERANCE);
        }

        void TestPseudoInverseLarge()
        {
            constexpr int M = 61250;
            constexpr int N = 350;

            Eigen::MatrixXd m1 = Eigen::MatrixXd::Random(M, N);
            Eigen::MatrixXd m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, cuda::JobType::S);
            
            //Note: for large matrices the smaller intermediate matrix is required to avoid std::bad_alloc issues
            Eigen::MatrixXd calculatedm1 = m1 * (m1d * m1);
            ASSERT_MEQD(m1, calculatedm1, TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(N,N), m1d * m1, TOLERANCE);
        }

        void TestPseudoInverseSKA()
        {
            constexpr int M = 130817; // TODO(calgray): cudamalloc
            constexpr int N = 512;

            Eigen::MatrixXd m1 = Eigen::MatrixXd::Random(M, N);
            Eigen::MatrixXd m1d = icrar::cuda::PseudoInverse(m_cusolverDnCtx, m1, cuda::JobType::S);
            
            //Note: for large matrices the smaller intermediate matrix is required to avoid memory issues
            Eigen::MatrixXd calculatedm1 = m1 * (m1d * m1);
            ASSERT_MEQD(m1, calculatedm1, TOLERANCE);
            ASSERT_MEQD(Eigen::MatrixXd::Identity(N,N), m1d * m1, TOLERANCE);
        }
    };

    TEST_F(CudaMatrixEigenTests, TestGpuVectorAdd10) { TestVectorAdd(); }
    TEST_F(CudaMatrixEigenTests, DISABLED_TestGpuPseudoInverse23A) { TestPseudoInverse23(cuda::JobType::A); }
    TEST_F(CudaMatrixEigenTests, DISABLED_TestGpuPseudoInverse23S) { TestPseudoInverse23(cuda::JobType::S); }
    TEST_F(CudaMatrixEigenTests, DISABLED_TestGpuPseudoInverse32Degenerate) { TestPseudoInverse32Degenerate(); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse32A) { TestPseudoInverse32(cuda::JobType::A); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse32S) { TestPseudoInverse32(cuda::JobType::S); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse33A) { TestPseudoInverse33(cuda::JobType::A); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse33S) { TestPseudoInverse33(cuda::JobType::S); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse42A) { TestPseudoInverse42(cuda::JobType::A); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverse42S) { TestPseudoInverse42(cuda::JobType::S); }

    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverseMWA) { TestPseudoInverseMWA(cuda::JobType::S); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverseLarge) { TestPseudoInverseLarge(); }
    TEST_F(CudaMatrixEigenTests, TestGpuPseudoInverseSKA) { TestPseudoInverseSKA(); }
}
