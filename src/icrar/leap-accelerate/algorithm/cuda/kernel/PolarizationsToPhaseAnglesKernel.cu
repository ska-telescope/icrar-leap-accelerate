#include "hip/hip_runtime.h"
/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 * MA 02111 - 1307  USA
 */

#include "RotateVisibilitiesKernel.h"
#include <icrar/leap-accelerate/exception/exception.h>

namespace icrar
{
namespace cuda
{
    __global__ void g_AvgDataToPhaseAngles(
        const Eigen::Map<const Eigen::VectorXi> I1,
        const Eigen::Map<const Eigen::Matrix<thrust::complex<double>, -1, -1>> avgData,
        Eigen::Map<Eigen::VectorXd> phaseAnglesI1)
    {
        int row = blockDim.x * blockIdx.x + threadIdx.x;
        if(row < I1.rows())
        {
            phaseAnglesI1(row) = thrust::arg(avgData(I1(row), 0));
        }
    }

    __host__ void AvgDataToPhaseAngles(const device_vector<int>& I1, const device_matrix<std::complex<double>>& avgData, device_vector<double>& phaseAnglesI1)
    {
        if(I1.GetRows()+1 != phaseAnglesI1.GetRows())
        {
            throw invalid_argument_exception("incorrect number of columns", "phaseAnglesI1", __FILE__, __LINE__);
        }

        dim3 blockSize = dim3(1024, 1, 1);
        dim3 gridSize = dim3(static_cast<int>(ceil(static_cast<double>(I1.GetRows()) / blockSize.x)), 1, 1);

        using MatrixXcd = Eigen::Matrix<thrust::complex<double>, -1, -1>;
        auto I1Map = Eigen::Map<const Eigen::VectorXi>(I1.Get(), I1.GetRows());
        auto avgDataMap = Eigen::Map<const MatrixXcd>((thrust::complex<double>*)avgData.Get(), avgData.GetRows(), avgData.GetCols());
        auto phaseAnglesI1Map = Eigen::Map<Eigen::VectorXd>(phaseAnglesI1.Get(), phaseAnglesI1.GetRows());
        g_AvgDataToPhaseAngles<<<blockSize, gridSize>>>(I1Map, avgDataMap, phaseAnglesI1Map);
    }
} // namespace cuda
} // namespace icrar
