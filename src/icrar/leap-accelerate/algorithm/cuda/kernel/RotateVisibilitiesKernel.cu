#include "hip/hip_runtime.h"
/**
 * ICRAR - International Centre for Radio Astronomy Research
 * (c) UWA - The University of Western Australia
 * Copyright by UWA(in the framework of the ICRAR)
 * All rights reserved
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 * MA 02111 - 1307  USA
 */

#include "RotateVisibilitiesKernel.h"
#include <icrar/leap-accelerate/math/cuda/math.cuh>
#include <icrar/leap-accelerate/math/cpu/math.h>


namespace icrar
{
namespace cuda
{
    template<typename T>
    constexpr int64_t CeilDiv(int x, int y)
    {
        return (x + y - T(1)) / y;
    }

    /**
     * @brief Rotates visibilities in parallel for baselines and channels
     * @note Atomic operator required for writing to @p pAvgData
     * 
     * @param constants measurement set constants
     * @param dd direction dependent rotation 
     * @param UVW unrotated uvws
     * @param integrationData inout integration data 
     * @param rotAvgVis output rotAvgVis to increment
     */
    __global__ void g_RotateVisibilities(
        const icrar::cpu::Constants constants,
        const Eigen::Matrix3d dd,
        const Eigen::Map<const Eigen::Matrix<double, 3, Eigen::Dynamic>> UVWs,
        Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 4>> integrationData,
        Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 2>> rotAvgVis);

    __host__ void RotateVisibilities(DeviceIntegration& integration, DeviceMetaData& metadata)
    {
        const auto& constants = metadata.GetConstants(); 
        assert(constants.channels == integration.GetChannels() && integration.GetChannels() == integration.GetVis().GetDimensionSize(2));
        assert(constants.nbaselines == metadata.GetAvgData().GetRows() && integration.GetRows() == integration.GetVis().GetDimensionSize(1));
        assert(constants.num_pols == integration.GetVis().GetDimensionSize(0));

        auto integrationDataMap = Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 4>>(
            reinterpret_cast<hipDoubleComplex*>(integration.GetVis().Get()),
            static_cast<int>(integration.GetVis().GetDimensionSize(0)), // inferring (const int) causes error
            static_cast<int>(integration.GetVis().GetDimensionSize(1)), // inferring (const int) causes error
            static_cast<int>(integration.GetVis().GetDimensionSize(2)), // inferring (const int) causes error
            static_cast<int>(integration.GetVis().GetDimensionSize(3)) // inferring (const int) causes error
        );

        auto rotAvgVisMap = Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 2>>(
            reinterpret_cast<hipDoubleComplex*>(metadata.GetAvgData().Get()),
            static_cast<int>(metadata.GetAvgData().GetRows()), // inferring (const int) causes error
            static_cast<int>(metadata.GetAvgData().GetCols()) // inferring (const int) causes error
        );

        const auto UVWMap = Eigen::Map<const Eigen::Matrix<double, 3, Eigen::Dynamic>>(
            reinterpret_cast<const double*>(metadata.GetUVW().Get()),
            3,
            metadata.GetUVW().GetCount()
        );

        dim3 blockSize = dim3(8, 128, 1); // block size can be any value where the product is <=1024
        dim3 gridSize = dim3(
            cpu::ceil_div<int64_t>(integration.GetNumChannels(), blockSize.x),
            cpu::ceil_div<int64_t>(integration.GetNumBaselines(), blockSize.y),
            cpu::ceil_div<int64_t>(integration.GetNumTimesteps(), blockSize.z)
        );
        g_RotateVisibilities<<<gridSize, blockSize>>>(
            constants,
            metadata.GetDD(),
            UVWMap,
            integrationDataMap,
            rotAvgVisMap);
        checkCudaErrors(hipGetLastError());
    }

    __global__ void g_RotateVisibilities(
        const icrar::cpu::Constants constants,
        const Eigen::Matrix3d dd,
        const Eigen::Map<const Eigen::Matrix<double, 3, Eigen::Dynamic>> UVWs,
        Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 4>> integrationData,
        Eigen::TensorMap<Eigen::Tensor<hipDoubleComplex, 2>> rotAvgVis)
    {
        const int integration_polarizations = integrationData.dimension(0);
        const int integration_channels = integrationData.dimension(1);
        const int integration_baselines = integrationData.dimension(2);
        const int integration_timesteps = integrationData.dimension(3);

        constexpr double two_pi = 2 * HIP_PI;

        //parallel execution per channel
        int channel = blockDim.x * blockIdx.x + threadIdx.x;
        int baseline = blockDim.y * blockIdx.y + threadIdx.y;
        int timestep = blockDim.z * blockIdx.z + threadIdx.z;
        int row = baseline + (integration_baselines * timestep);

        if(baseline < integration_baselines && channel < integration_channels)
        {
            // Rotation
            Eigen::Vector3d rotatedUVW = dd * UVWs.col(baseline); //TODO: use row 
            double shiftFactor = -two_pi * (rotatedUVW.z() - UVWs.col(row).z()); //TODO: use row
            double shiftRad = shiftFactor / constants.GetChannelWavelength(channel);
            hipDoubleComplex shiftCoeff = cuCexp(make_hipDoubleComplex(0.0, shiftRad));
            for(int polarization = 0; polarization < integration_polarizations; polarization++)
            {
                integrationData(polarization, channel, baseline, timestep)
                = hipCmul(integrationData(polarization, channel, baseline, timestep), shiftCoeff);
            }

            // Averaging
            bool hasNaN = false;
            for(int polarization = 0; polarization < integration_polarizations; polarization++)
            {
                hipDoubleComplex n = integrationData(polarization, channel, baseline, timestep);
                hasNaN |= isnan(n.x) || isnan(n.y);
            }
            if(!hasNaN)
            {
                // XX + YY
                atomicAdd(&rotAvgVis(baseline, 0).x, integrationData(0, channel, baseline, timestep).x);
                atomicAdd(&rotAvgVis(baseline, 0).y, integrationData(0, channel, baseline, timestep).y);
                atomicAdd(&rotAvgVis(baseline, 0).x, integrationData(integration_polarizations - 1, channel, baseline, timestep).x);
                atomicAdd(&rotAvgVis(baseline, 0).y, integrationData(integration_polarizations - 1, channel, baseline, timestep).y);
            }
        }
    }
} // namespace cuda
} // namespace icrar
